#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1618
#define T 1024 

__global__ void vecAdd (int *in) {
	int me = blockIdx.x * blockDim.x + threadIdx.x;
	in[me] += 1;
	// printf("me: %d\n", me); 
}


int main(int argc, char** argv) {
	// int k = atoi(argv[1]); 
	int n = atoi(argv[1]); 

	int *h_x; // host array
	int *h_result; // host result
	int *d_x; // device array
	int *d_result; // device result

	// size_t arr_size = n * sizeof(int); 	// size of array 
	h_x = (int*) malloc(n * sizeof(int)); 		// allocate memory for host input
	h_result = (int*) malloc(n * sizeof(int));  // allocate memory on host result
	hipMalloc((void**) &d_x, n * sizeof(int));	// allocate memory for device
	hipMalloc((void**) &d_result, n * sizeof(int)); // allocate memory for device result

	for (int i = 0; i < n; i++) {
		h_x[i] = i;
	}

	// copy host input to device input
	hipMemcpy(d_x, h_x, n * sizeof(int),hipMemcpyHostToDevice);

	// allocate blocksizes and gridsizes
	dim3 gridDim(1, 1, 1); 
	dim3 blockDim(n, 1, 1); 

	// increment 1 to d_x write to d_result
	vecAdd<<<gridDim, blockDim>>>(d_x);//, d_result);

	// copy from device result to host result
	hipMemcpy(h_result, d_x, n * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		printf("h_result: %d: %d\n", i, h_result[i]);
	}

	// free memory of GPU
	hipFree(d_x);
	hipFree(d_result);
}

