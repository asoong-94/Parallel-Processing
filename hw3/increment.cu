#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void increment(float *x, float *y) {
	int me = threadIdx.x;
	y[me] = x[me] += 1;
	__syncthreads();
}

int main(int argc, char** argv) {
	int n = 10;

	// arrays for host 
	float *h_input; 
	float *h_output; 

	// allocate space on host
	h_input = (float*)malloc(n * sizeof(float));
	h_output = (float*)malloc(n * sizeof(float));


	// arrays for device 
	float *d_input; 
	float *d_output; 

	// allocate space on device
	hipMalloc((void**) &d_input, n * sizeof(float)); 
	hipMalloc((void**) &d_output, n * sizeof(float)); 

	// populate host array
	for (int i = 0; i < n; i++) {
		h_input[i] = i;
	}

	// copy host input to device input array 
	hipMemcpy(d_input, h_input, n * sizeof(float), hipMemcpyHostToDevice);

	dim3 gridDim(1,1);
	dim3 gridBlock(n, 1);

	// invoke the increment kernel
	increment<<< gridDim, gridBlock >>>(d_input, d_output);

	// copy device output back to host output
	hipMemcpy(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost);

	// print result 
	for (int i = 0; i < n; i++) {
		printf("%f\n", h_output[i]);
	}

	// free memory on GPU
	hipFree(d_output); 
	hipFree(d_input); 

	return 0;
}

