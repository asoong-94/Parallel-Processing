#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



#define MAX_THREADS_PER_BLOCK 1024 
#define RANDMAX 100

// __global__ void mean(float **subarr, int start, int end, float *out) {
// 	float tot = 0; 
// 	// for (int i = 0; i < end; i++) {
// 	// 	tot += *subarr[i]; 
// 	// }

// 	for (int i = 0; i < 10; i++0) {
// 	}
// 	__syncthreads(); 

// }

// __global__ void kernel(float *x) {
// 	// vsize
// 	extern __shared__ float sv[];
// 	int me = blockIdx.x * threadIdx.x; 



// }



// void maxburst(float *x, int n, int k, int *startend, float *bigmax) {
// 	int NUM_BLOCKS = n / MAX_THREADS_PER_BLOCK;
// 	int vsize = n * sizeof(float);

// 	float *d_x; // device array 
// 	cudaMalloc((void**) &d_x, vsize); // allocate memory on device 
// 	cudaMemcpy(d_x, x, vsize, cudaMemcpyHostToDevice);

// 	dim3 gridDim(NUM_BLOCKS,1); 
// 	dim3 blockDim(MAX_THREADS_PER_BLOCK,1,1);
	
// 	//kernel<<< gridDim, blockDim >>>(d_x);

// 	float res = 0;
// 	float res_out = 0;	
// 	cudaMalloc((void**) &res, sizeof(float));
// 	mean<<<gridDim, blockDim>>>(&d_x, 10, 15, res); 
// 	cudaMemcpy(&res_out, &res, sizeof(float), cudaMemcpyDeviceToHost);
// 	printf("mean: %f", res_out);
// }


__global__ void findMax(float *d_in, float *d_out) {

	int me = blockDim.x * blockIdx.x + threadIdx.x;
}


int main(int argc, char** argv) {
	int n = atoi(argv[1]); // array input size
	int k = atoi(argv[2]); // window size

	// size of input array
	int input_size = n * sizeof(float); 

	// host input and output array
	float *h_in;
	float *h_out; 

	// allocate memory for host arrays 
	h_in = (float*)malloc(input_size);
	h_out = (float*)malloc(input_size);

	// fill host array with random numbers 
	for (int i = 0; i < n; i++) {
		h_in[i] = rand() / (float)RANDMAX; 
	}

	// device input and output array 
	float *d_in; 
	float *d_out; 

	// allocate memory for device arrays 
	hipMalloc((void**) &d_in, input_size);
	hipMalloc((void**) &d_out, input_size);

	// create kernel invocation parameters
	int NUM_BLOCKS = n / MAX_THREADS_PER_BLOCK;
	dim3 dimGrid(NUM_BLOCKS, 1); 
	dim3 dimBlock(MAX_THREADS_PER_BLOCK, 1);


	findMax <<< dimGrid, dimBlock >>>(d_in, d_out); 



	return 0; 
}




