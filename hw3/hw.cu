#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



#define MAX_THREADS_PER_BLOCK 1024 
#define RANDMAX 100

// __global__ void mean(float **subarr, int start, int end, float *out) {
// 	float tot = 0; 
// 	// for (int i = 0; i < end; i++) {
// 	// 	tot += *subarr[i]; 
// 	// }

// 	for (int i = 0; i < 10; i++0) {
// 	}
// 	__syncthreads(); 

// }

// __global__ void kernel(float *x) {
// 	// vsize
// 	extern __shared__ float sv[];
// 	int me = blockIdx.x * threadIdx.x; 



// }



// void maxburst(float *x, int n, int k, int *startend, float *bigmax) {
// 	int NUM_BLOCKS = n / MAX_THREADS_PER_BLOCK;
// 	int vsize = n * sizeof(float);

// 	float *d_x; // device array 
// 	cudaMalloc((void**) &d_x, vsize); // allocate memory on device 
// 	cudaMemcpy(d_x, x, vsize, cudaMemcpyHostToDevice);

// 	dim3 gridDim(NUM_BLOCKS,1); 
// 	dim3 blockDim(MAX_THREADS_PER_BLOCK,1,1);
	
// 	//kernel<<< gridDim, blockDim >>>(d_x);

// 	float res = 0;
// 	float res_out = 0;	
// 	cudaMalloc((void**) &res, sizeof(float));
// 	mean<<<gridDim, blockDim>>>(&d_x, 10, 15, res); 
// 	cudaMemcpy(&res_out, &res, sizeof(float), cudaMemcpyDeviceToHost);
// 	printf("mean: %f", res_out);
// }

__global__ void findMax(float *x, int n, int k, int *startend, float *bigmax) {
	// shared memory of size: input_size
	extern __shared__ float s[];

	// thread id
	int me;  

	// copy global data to shared data 
	s[me] = x[me];

	int perstart; // period start
	int perlen;   // period length 
	int perend;   // perlen end 
	int pl1;      // perlen - 1

	// for best found by this thread so far 
	int mystartmax; // start location 
	int myendmax;   // end location 

	float mymaxval; // max value 

	float xbar; 	// scratch variable 

	me = blockDim.x * blockIdx.x + threadIdx.x;
	mymaxval = -1;







	__syncthreads();
}


int main(int argc, char** argv) {
	int n = atoi(argv[1]); // array input size
	int k = atoi(argv[2]); // window size

	// size of input array
	int input_size = n * sizeof(float); 

	// host input and output array
	float *h_in;

	// allocate memory for host arrays 
	h_in = (float*)malloc(input_size);

	// fill host array with random numbers 
	for (int i = 0; i < n; i++) {
		h_in[i] = rand() / (float)RANDMAX; 
	}

	// device input and output array 
	float *d_in; 

	// allocate memory for device arrays 
	hipMalloc((void**) &d_in, input_size);

	// copy host input to device input 
	hipMemcpy(d_in, h_in, input_size, hipMemcpyHostToDevice);

	// other host parameters
	int *h_startend; 
	h_startend = (int*) malloc(2 * sizeof(int));
	float *h_bigmax = 0;

	// other device parameters 
	int *d_startend;
	hipMalloc((void**) &d_startend, 2 * sizeof(int));
	float *d_bigmax = 0;

	// create kernel invocation parameters
	int NUM_BLOCKS = n / MAX_THREADS_PER_BLOCK;
	dim3 dimGrid(NUM_BLOCKS, 1); 
	dim3 dimBlock(MAX_THREADS_PER_BLOCK, 1);

	// kernel invocation
	findMax <<< dimGrid, dimBlock, input_size >>>(d_in, n, k, d_startend , d_bigmax); 

	// copy result from device to host
	hipMemcpy(d_startend, h_startend, input_size, hipMemcpyDeviceToHost);
	hipMemcpy(d_bigmax, h_bigmax, input_size, hipMemcpyDeviceToHost);


	// free gpu memory 
	hipFree(d_startend); 
	hipFree(d_bigmax); 

	return 0; 
}



