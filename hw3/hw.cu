#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



#define MAX_THREADS_PER_BLOCK 1024 
#define RANDMAX 100

__device__ float mean(float *y, int s, int e) {
	int i; 
	float total = 0; 
	for (int i = s; i < e; i++) {
		total += y[i];
	}
	return total / (e - s + 1);
}

__global__ void findMax(float *x, int n, int k, int *startend, float *bigmax) {
	// shared memory of size: input_size
	extern __shared__ float s[];

	// thread id
	int me = blockDim.x * blockIdx.x + threadIdx.x;
	printf("me: %d", me);
	// copy global data to shared data 
	s[me] = x[me];

	int perstart; // period start
	int perlen;   // period length 
	int perend;   // perlen end 
	int pl1;      // perlen - 1

	// for best found by this thread so far 
	int mystartmax; // start location 
	int myendmax;   // end location 

	float mymaxval; // max value 

	float xbar; 	// scratch variable 

	mymaxval = -1;

	for (perstart = me; perstart <= n - k; perstart ++) {
		for (perlen = k; perlen <= n - perstart; perlen ++) {
			if (perlen == k) {
				xbar = mean(s, perstart, perend); 
			}
			else {
				// update hold mean 
				pl1 = perlen - 1; 
				xbar = (pl1 * xbar + s[perend]) / perlen; 
			}

			if (xbar > mymaxval) {
				mymaxval = xbar; 
				mystartmax = perstart; 
				myendmax = perend; 
			}
		}
	}

	__syncthreads();
	if (mymaxval > *bigmax) {
		*bigmax = mymaxval; 
		startend[0] = mystartmax; 
		startend[1] = myendmax; 
	}
}


int main(int argc, char** argv) {
	int n = atoi(argv[1]); // array input size
	int k = atoi(argv[2]); // window size

	// size of input array
	int input_size = n * sizeof(float); 

	// host input and output array
	float *h_in;

	// allocate memory for host arrays 
	h_in = (float*)malloc(input_size);

	// fill host array with random numbers 
	for (int i = 0; i < n; i++) {
		h_in[i] = rand() / (float)RANDMAX; 
	}

	// device input and output array 
	float *d_in; 

	// allocate memory for device arrays 
	hipMalloc((void**) &d_in, input_size);

	// copy host input to device input 
	hipMemcpy(d_in, h_in, input_size, hipMemcpyHostToDevice);

	// other host parameters
	int *h_startend; 
	h_startend = (int*) malloc(2 * sizeof(int));
	float *h_bigmax = 0;

	// other device parameters 
	int *d_startend;
	hipMalloc((void**) &d_startend, 2 * sizeof(int));
	float *d_bigmax = 0;

	// create kernel invocation parameters
	int NUM_BLOCKS = n / MAX_THREADS_PER_BLOCK;
	dim3 dimGrid(NUM_BLOCKS, 1); 
	dim3 dimBlock(MAX_THREADS_PER_BLOCK, 1);

	// kernel invocation
	findMax <<< dimGrid, dimBlock, input_size >>>(d_in, n, k, d_startend , d_bigmax); 

	// copy result from device to host
	hipMemcpy(d_startend, h_startend, input_size, hipMemcpyDeviceToHost);
	hipMemcpy(d_bigmax, h_bigmax, input_size, hipMemcpyDeviceToHost);

	printf("dstartend.1: %d\n", h_startend[0]);
	printf("dstartend.2: %d\n", h_startend[1]);
	printf("d_bigmax: %f\n", h_bigmax);

	// free gpu memory 
	hipFree(d_startend); 
	hipFree(d_bigmax); 

	return 0; 
}




