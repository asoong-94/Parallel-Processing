#include "hip/hip_runtime.h"
#include <iostream> 
#include <math.h>

void add (int n, float *x, float *y) {
	for (int i = 0; i < n; i++) {
		y[i] = x[i] + y[i];
	}
}

__global__ void add(int n, float *x, float *y) {
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main() {
	long N  = 1 << 30; 
	// float *x = new float[N];
	// float *y = new float[N];
	float *x; 
	float *y; 
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));



	for (int i = 0; i < N; i++) {
		x[i] = 1.0f; 
		y[i] = 2.0f; 
	}

	// add(N, x, y); 

	add <<<1, 1>>>(N, x, y);
	 // Check for errors (all values should be 3.0f)
	
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
	maxError = fmax(maxError, fabs(y[i]-3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	delete []x;
	delete []y;

	return 0;
}